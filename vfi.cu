#include "hip/hip_runtime.h"
#define nk 2500
#define nz 23
#define tol 1e-7
#define maxiter 2500
#define kwidth 1.5

/* Includes, system */
#include <fstream>
#include <iostream>
#include <iomanip>
#include <string>

// Includes, Thrust
#include <thrust/for_each.h>
#include <thrust/extrema.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>

// Includes, cuda
#include <hipblas.h>
#include "cuda_helpers.h"

// Includes, my own creation
#include "common.h"

// Includes model stuff
#include "rbcmodel.h"


/// This function finds the value of RHS given k', k, z
__host__ __device__
double rhsvalue (state s, int i_z, double kplus, int i_kplus, double* EV, para p) {
	return log(s.z*pow(s.k,p.ttheta)+(1-p.ddelta)*s.k-kplus) + p.bbeta*EV[i_kplus+i_z*nk];
};

// This find the max using binary search and assumes concavity
__host__ __device__
void concavemax(double k, double z, const int left_ind, const int right_ind, const int i_k,const int i_z, double* K, double* EV, int* koptind, double* Vplus, para p) {
	int index = i_k + i_z*nk;

	if (right_ind-left_ind==1) {
		double left_value, right_value;
		left_value = rhsvalue(state(k,z,p),i_z,K[left_ind],left_ind, EV, p);
		right_value = rhsvalue(state(k,z,p),i_z,K[right_ind],right_ind, EV, p);
		if (left_value>right_value) {
			Vplus[index] = left_value;
			koptind[index] = left_ind;
		} else {
			Vplus[index] = right_value;
			koptind[index] = right_ind;
		};
	} else if (right_ind-left_ind==2) {
		double value1 = rhsvalue(state(k,z,p),i_z,K[left_ind],left_ind, EV, p);
		double value2 = rhsvalue(state(k,z,p),i_z,K[left_ind+1],left_ind+1, EV, p);
		double value3 = rhsvalue(state(k,z,p),i_z,K[right_ind],right_ind, EV, p);
		if (value1 < value2) {
			if (value2 < value3) {
				Vplus[index] = value3;
				koptind[index] = right_ind;
			} else {
				Vplus[index] = value2;
				koptind[index] = left_ind+1;
			}
		} else {
			if (value1 < value3) {
				Vplus[index] = value3;
				koptind[index] = right_ind;
			} else {
				Vplus[index] = value1;
				koptind[index] = left_ind;
			}
		}
	} else {
		int ind1 = left_ind; int ind4 = right_ind;
		int ind2, ind3;
		double value1, value2, value3;
		while (ind4 - ind1 > 2) {
			ind2 = (ind1+ind4)/2;
			ind3 = ind2 + 1;
			value2 = rhsvalue(state(k,z,p),i_z,K[ind2],ind2, EV, p);
			value3 = rhsvalue(state(k,z,p),i_z,K[ind3],ind3, EV, p);
			if (value2 < value3) {
				ind1 = ind2;
			} else {
				ind4 = ind3;
			};
		};

		// Now the number of candidates is reduced to three
		value1 = rhsvalue(state(k,z,p),i_z,K[ind1],ind1, EV, p);
		value2 = rhsvalue(state(k,z,p),i_z,K[ind4-1],ind4-1, EV, p);
		value3 = rhsvalue(state(k,z,p),i_z,K[ind4],ind4, EV, p);

		if (value1 < value2) {
			if (value2 < value3) {
				Vplus[index] = value3;
				koptind[index] = ind4;
			} else {
				Vplus[index] = value2;
				koptind[index] = ind4-1;
			}
		} else {
			if (value1 < value3) {
				Vplus[index] = value3;
				koptind[index] = ind4;
			} else {
				Vplus[index] = value1;
				koptind[index] = ind1;
			}
		}
	}
};

// This functor optimal kplus and Vplus
struct kplusVplusopt
{
	// Data member
	double *K, *Z, *EV;
	int *koptind;
    double *Vplus;
	para p;

	// Construct this object, create util from _util, etc.
	__host__ __device__
    kplusVplusopt(double* K_ptr, double* Z_ptr, double* EV_ptr, int* koptind_ptr, double* Vplus_ptr, para _p) {
		K = K_ptr; Z = Z_ptr; EV = EV_ptr;
		koptind = koptind_ptr; Vplus = Vplus_ptr;
		p = _p;
	};

	__host__ __device__
	void operator()(int index) {
		// Perform ind2sub
		int subs[2];
		int size_vec[2];
		size_vec[0] = nk;
		size_vec[1] = nz;
		ind2sub(2,size_vec,index,subs);
		int i_k = subs[0];
		int i_z = subs[1];

		// Find and construct state and control, otherwise they won't update in the for loop
		double k =K[i_k]; double z=Z[i_z];

		// Exploit concavity to update V
		concavemax(k, z, 0, nk-1, i_k, i_z, K, EV, koptind, Vplus, p);

	};
};

// This functor calculates the distance
struct myDist {
	// Tuple is (V1low,Vplus1low,V1high,Vplus1high,...)
	template <typename Tuple>
		__host__ __device__
	double operator()(Tuple t)
	{
		return abs(thrust::get<0>(t)-thrust::get<1>(t));
	}
};

int main(int argc, char ** argv)
{
	// Initialize Parameters
	para p; // in #include "rbcmodel.h"

	// Set Model Parameters
	p.bbeta = 0.9825;
	p.ddelta = 0.025;
	p.ttheta = 0.36;
	p.zbar = 1.0;
	p.rrhozz = 0.9457;
	p.std_epsz = 0.0045*0.0045;
	p.complete(); // complete all implied para, find S-S

	std::cout << std::setprecision(16) << "kss: " << p.kss << std::endl;
	std::cout << std::setprecision(16) << "zss: " << p.zbar << std::endl;
	std::cout << std::setprecision(16) << "tol: " << tol << std::endl;

	// Select Device
	// int num_devices;
	// hipGetDeviceCount(&num_devices);
	if (argc > 1) {
		int gpu = atoi(argv[1]);
		hipSetDevice(gpu);
	};
	// Only for cuBLAS
	const double alpha = 1.0;
	const double beta = 0.0;

	// Create all STATE, SHOCK grids here
	h_vec_d h_K(nk);
	h_vec_d h_Z(nz);
    h_vec_d h_logZ(nz);
	h_vec_d h_V(nk*nz,0.0);
	h_vec_d h_Vplus(nk*nz,0);
	h_vec_i h_koptind(nk*nz);
	h_vec_d h_EV(nk*nz,0.0);
	h_vec_d h_P(nz*nz, 0);

    load_vec(h_V,"./results/Vgrid.csv"); // in #include "cuda_helpers.h"

	// Create capital grid
	double minK = 1.0/kwidth*p.kss;
	double maxK = kwidth*p.kss;
	linspace(minK,maxK,nk,thrust::raw_pointer_cast(h_K.data())); // in #include "cuda_helpers.h"

	// Create shocks grids
	h_vec_d h_shockgrids(nz);
	double* h_logZ_ptr = thrust::raw_pointer_cast(h_logZ.data());
	double* h_P_ptr = thrust::raw_pointer_cast(h_P.data());
    	tauchen(p.rrhozz, p.std_epsz, h_logZ_ptr, h_P_ptr); // in #include "cuda_helpers.h"
	for (int i_shock = 0; i_shock < nz; i_shock++) {
		h_Z[i_shock] = p.zbar*exp(h_logZ[i_shock]);
	};

	// Copy to the device
	d_vec_d d_K = h_K;
	d_vec_d d_Z = h_Z;
	d_vec_d d_V = h_V;
	d_vec_d d_Vplus = h_Vplus;
	d_vec_i d_koptind = h_koptind;
	d_vec_d d_EV = h_EV;
	d_vec_d d_P = h_P;

	// Obtain device pointers to be used by cuBLAS
	double* d_K_ptr = raw_pointer_cast(d_K.data());
	double* d_Z_ptr = raw_pointer_cast(d_Z.data());
	double* d_V_ptr = raw_pointer_cast(d_V.data());
	double* d_Vplus_ptr = raw_pointer_cast(d_Vplus.data());
	int* d_koptind_ptr = raw_pointer_cast(d_koptind.data());
	double* d_EV_ptr = raw_pointer_cast(d_EV.data());
	double* d_P_ptr = raw_pointer_cast(d_P.data());

	// Firstly a virtual index array from 0 to nk*nk*nz
	thrust::counting_iterator<int> begin(0);
	thrust::counting_iterator<int> end(nk*nz);

    // Create Timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    // Start Timer
	hipEventRecord(start,NULL);

	// Step.1 Has to start with this command to create a handle
	hipblasHandle_t handle;

	// Step.2 Initialize a cuBLAS context using Create function,
	// and has to be destroyed later
	hipblasCreate(&handle);

	double diff = 10;  int iter = 0;
	while ((diff>tol)&&(iter<maxiter)){
		// Find EMs for low and high
		hipblasDgemm(handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_T,
			nk, nz, nz,
			&alpha,
			d_V_ptr,
			nk,
			d_P_ptr,
			nz,
			&beta,
			d_EV_ptr,
			nk);

		// Directly find the new Value function
		thrust::for_each(
			begin,
			end,
			kplusVplusopt(d_K_ptr, d_Z_ptr, d_EV_ptr, d_koptind_ptr, d_Vplus_ptr, p)
                         );

		// Find diff
		diff = thrust::transform_reduce(
			thrust::make_zip_iterator(thrust::make_tuple(d_V.begin(),d_Vplus.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(d_V.end()  ,d_Vplus.end())),
			myDist(),
			0.0,
			thrust::maximum<double>()
			);

// maximum in #include <thrust/extrema.h>

		std::cout << "diff is: "<< diff << std::endl;

		// update correspondence
		d_V = d_Vplus;

		std::cout << ++iter << std::endl;
		std::cout << "=====================" << std::endl;

	};

	//==========cuBLAS stuff ends=======================
	// Step.3 Destroy the handle.
	hipblasDestroy(handle);

	// Stop Timer
	hipEventRecord(stop,NULL);
	hipEventSynchronize(stop);
	float msecTotal = 0.0;
	hipEventElapsedTime(&msecTotal, start, stop);

	// Compute and print the performance
	float msecPerMatrixMul = msecTotal;
	std::cout << "Time= " << msecPerMatrixMul << " msec, iter= " << iter << std::endl;

	// Copy back to host and print to file
	h_V = d_V;
	h_EV = d_EV;
	h_koptind = d_koptind;

    save_vec(h_K,"./results/Kgrid.csv"); // in #include "cuda_helpers.h"
	save_vec(h_Z,"./results/Zgrid.csv"); // in #include "cuda_helpers.h"
	save_vec(h_P,"./results/Pgrid.csv"); // in #include "cuda_helpers.h"
	save_vec(h_V,"./results/Vgrid.csv"); // in #include "cuda_helpers.h"
	std::cout << "Policy functions output completed." << std::endl;

	// Export parameters to MATLAB
	p.exportmatlab("./MATLAB/vfi_para.m");

	return 0;
}
